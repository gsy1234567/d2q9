#include "hip/hip_runtime.h"
#include "array3d.cuh"
#include "types.cuh"
#include <iostream>
#include <cstdlib>
#include <hip/hip_cooperative_groups.h>
#include <fstream>
#include<iomanip>



using namespace cooperative_groups;
using namespace gsy_cuda;

__constant__ u32 dev_obstacles [0x4000];
u32 host_obstacles[0x4000];

struct cpu_data_t {
    float *speeds;
};

struct gpu_data_t {
    array3d<float, Device> dev_speed_src;
    array3d<float, Device> dev_speed_dst;
    float * dev_inlets;
    u32 obstacles_pitch;
};

__device__ bool check_obstacle(u32 pitch, u32 x_idx, u32 y_idx) {
    return dev_obstacles[y_idx * pitch + (x_idx >> 5)] & (1U << (x_idx & 31U));
}

__host__ bool check_obstacle_host(u32 pitch, u32 x_idx, u32 y_idx) {
    return host_obstacles[y_idx * pitch + (x_idx >> 5)] & (1U << (x_idx & 31U));
}

__host__ void set_obstacle(u32 pitch, u32 x_idx, u32 y_idx) {
    host_obstacles[y_idx * pitch + (x_idx >> 5)] |= (1U << (x_idx & 31U));
}

__device__ void load_speeds(u32 x_idx, u32 y_idx, float* local_tmp, array3d<float, Device> dev_speed_src) {
    local_tmp[0] = dev_speed_src.at(x_idx, y_idx, 0);
    local_tmp[1] = dev_speed_src.at(x_idx, y_idx, 1);
    local_tmp[2] = dev_speed_src.at(x_idx, y_idx, 2);
    local_tmp[3] = dev_speed_src.at(x_idx, y_idx, 3);
    local_tmp[4] = dev_speed_src.at(x_idx, y_idx, 4);
    local_tmp[5] = dev_speed_src.at(x_idx, y_idx, 5);
    local_tmp[6] = dev_speed_src.at(x_idx, y_idx, 6);
    local_tmp[7] = dev_speed_src.at(x_idx, y_idx, 7);
    local_tmp[8] = dev_speed_src.at(x_idx, y_idx, 8);
}

__device__ void collision(float* tmp, float omega) {

    constexpr float inv_c_sq = 3.f;
    constexpr float w0 = 4.f / 9.f;
    constexpr float w1 = 1.f / 9.f;
    constexpr float w2 = 1.f / 36.f;

    const float local_density = tmp[0] + tmp[1] + tmp[2] + tmp[3] + tmp[4] + tmp[5] + tmp[6] + tmp[7] + tmp[8];
    const float u_x = (tmp[1] + tmp[5] + tmp[8] - tmp[3] - tmp[6] - tmp[7]) / local_density;
    const float u_y = (tmp[2] + tmp[5] + tmp[6] - tmp[4] - tmp[7] - tmp[8]) / local_density;
    const float u_sq = u_x * u_x + u_y * u_y;
    const float tmp_term_0 = u_sq * (0.5f * inv_c_sq);
    //handle speed 0~2
    float ori_tmp_0 = tmp[0] * (1.f - omega);
    float ori_tmp_1 = tmp[1] * (1.f - omega);
    float ori_tmp_2 = tmp[2] * (1.f - omega);
    tmp[0] = 0.f * inv_c_sq;
    tmp[1] = u_x * inv_c_sq;
    tmp[2] = u_y * inv_c_sq;
    tmp[0] = w0 * local_density * (1.f + tmp[0] * (1.f + 0.5f * tmp[0]) - tmp_term_0);
    tmp[1] = w1 * local_density * (1.f + tmp[1] * (1.f + 0.5f * tmp[1]) - tmp_term_0);
    tmp[2] = w1 * local_density * (1.f + tmp[2] * (1.f + 0.5f * tmp[2]) - tmp_term_0);
    tmp[0] = omega * tmp[0] + ori_tmp_0;
    tmp[1] = omega * tmp[1] + ori_tmp_1;
    tmp[2] = omega * tmp[2] + ori_tmp_2;
    //handle speed 3~5
    ori_tmp_0 = tmp[3] * (1.f - omega);
    ori_tmp_1 = tmp[4] * (1.f - omega);
    ori_tmp_2 = tmp[5] * (1.f - omega);
    tmp[3] = u_x * -inv_c_sq;
    tmp[4] = u_y * -inv_c_sq;
    tmp[5] = (u_x + u_y) * inv_c_sq;
    tmp[3] = w1 * local_density * (1.f + tmp[3] * (1.f + 0.5f * tmp[3]) - tmp_term_0);
    tmp[4] = w1 * local_density * (1.f + tmp[4] * (1.f + 0.5f * tmp[4]) - tmp_term_0);
    tmp[5] = w2 * local_density * (1.f + tmp[5] * (1.f + 0.5f * tmp[5]) - tmp_term_0);
    tmp[3] = omega * tmp[3] + ori_tmp_0;
    tmp[4] = omega * tmp[4] + ori_tmp_1;
    tmp[5] = omega * tmp[5] + ori_tmp_2;
    //handle speed 6~8
    ori_tmp_0 = tmp[6] * (1.f - omega);
    ori_tmp_1 = tmp[7] * (1.f - omega);
    ori_tmp_2 = tmp[8] * (1.f - omega);
    tmp[6] = (-u_x + u_y) * inv_c_sq;
    tmp[7] = (u_x + u_y) * -inv_c_sq;
    tmp[8] = (u_x - u_y) * inv_c_sq;
    tmp[6] = w2 * local_density * (1.f + tmp[6] * (1.f + 0.5f * tmp[6]) - tmp_term_0);
    tmp[7] = w2 * local_density * (1.f + tmp[7] * (1.f + 0.5f * tmp[7]) - tmp_term_0);
    tmp[8] = w2 * local_density * (1.f + tmp[8] * (1.f + 0.5f * tmp[8]) - tmp_term_0);
    tmp[6] = omega * tmp[6] + ori_tmp_0;
    tmp[7] = omega * tmp[7] + ori_tmp_1;
    tmp[8] = omega * tmp[8] + ori_tmp_2;

}

__device__ void obstacle(float* tmp) {
    float tmp0 = tmp[1];
    float tmp1 = tmp[2];
    float tmp2 = tmp[5];
    float tmp3 = tmp[6];
    tmp[1] = tmp[3];
    tmp[2] = tmp[4];
    tmp[5] = tmp[7];
    tmp[6] = tmp[8];
    tmp[3] = tmp0;
    tmp[4] = tmp1;
    tmp[7] = tmp2;
    tmp[8] = tmp3;
} 

__device__ void save_speeds(u32 x_idx, u32 y_idx, const u32 nx, const u32 ny, const float* tmp, array3d<float, Device> dev_speed_dst) {
    //save speed0
    if(x_idx < nx - 1) {
        dev_speed_dst.at(x_idx, y_idx, 0) = tmp[0];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, y_idx, 0) = tmp[0];
        }
    }

    //save speed1
    if(x_idx < nx - 2) {
        dev_speed_dst.at(x_idx + 1, y_idx, 1) = tmp[1];
        if(x_idx == nx - 3) {
            dev_speed_dst.at(x_idx + 2, y_idx, 1) = tmp[1];
        }
    }

    //save speed2
    if(y_idx == 0 && x_idx < nx - 1) {
        dev_speed_dst.at(x_idx, 0, 2) = tmp[4];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, 0, 2) = tmp[4];
        }
    }
    if(x_idx < nx - 1 && y_idx < ny - 1) {
        dev_speed_dst.at(x_idx, y_idx + 1, 2) = tmp[2];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, y_idx + 1, 2) = tmp[2];
        }
    }

    //save speed3
    if(x_idx > 0) {
        dev_speed_dst.at(x_idx - 1, y_idx, 3) = tmp[3];
        if(x_idx == nx - 1) {
            dev_speed_dst.at(x_idx, y_idx, 3) = tmp[3];
        }
    }

    //save speed4
    if(y_idx == ny - 1 && x_idx < nx - 1) {
        dev_speed_dst.at(x_idx, y_idx, 4) = tmp[2];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, y_idx, 4) = tmp[2];
        }
    }
    if(x_idx < nx - 1 && y_idx > 0) {
        dev_speed_dst.at(x_idx, y_idx - 1, 4) = tmp[4];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, y_idx - 1, 4) = tmp[4];
        }
    }

    //save speed5
    if(y_idx == 0 && x_idx > 0 && x_idx < nx - 1) {
        dev_speed_dst.at(x_idx, 0, 5) = tmp[7];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, 0, 5) = tmp[7];
        }
    }
    if(x_idx < nx - 2 && y_idx < ny - 1) {
        dev_speed_dst.at(x_idx + 1, y_idx + 1, 5) = tmp[5];
        if(x_idx == nx - 3) {
            dev_speed_dst.at(x_idx + 2, y_idx + 1, 5) = tmp[5];
        }
    }

    //save speed6
    if(y_idx == 0 && x_idx < nx - 1) {
        dev_speed_dst.at(x_idx, 0, 6) = tmp[8];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, 0, 6) = tmp[8];
        }
    }
    if(x_idx > 0 && y_idx < ny - 1) {
        dev_speed_dst.at(x_idx - 1, y_idx + 1, 6) = tmp[6];
        if(x_idx == nx - 1) {
            dev_speed_dst.at(x_idx, y_idx + 1, 6) = tmp[6];
        }
    }

    //save speed7
    if(y_idx == ny - 1 && x_idx < nx - 1) {
        dev_speed_dst.at(x_idx, y_idx, 7) = tmp[5];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, y_idx, 7) = tmp[5];
        }
    }
    if(x_idx > 0 && y_idx > 0) {
        dev_speed_dst.at(x_idx - 1, y_idx - 1, 7) = tmp[7];
        if(x_idx == nx - 1) {
            dev_speed_dst.at(x_idx, y_idx - 1, 7) = tmp[7];
        }
    }

    //save speed8
    if(y_idx == ny - 1 && x_idx > 0 && x_idx < nx - 1) {
        dev_speed_dst.at(x_idx, y_idx, 8) = tmp[6];
        if(x_idx == nx - 2) {
            dev_speed_dst.at(x_idx + 1, y_idx, 8) = tmp[6];
        }
    }
    if(x_idx < nx - 2 && y_idx > 0) {
        dev_speed_dst.at(x_idx + 1, y_idx - 1, 8) = tmp[8];
        if(x_idx == nx - 3) {
            dev_speed_dst.at(x_idx + 2, y_idx - 1, 8) = tmp[8];
        }
    }
}

__device__ void left_boundary(u32 y_idx, array3d<float, Device> dev_speeds_dst, float* local_tmp, const float* dev_inlets) {
    constexpr float cst1 = 2.f / 3.f;
    constexpr float cst2 = 1.f / 6.f;
    constexpr float cst3 = 1.f / 2.f;
    local_tmp[0] = dev_speeds_dst.at(0, y_idx, 0);
    local_tmp[2] = dev_speeds_dst.at(0, y_idx, 2);
    local_tmp[3] = dev_speeds_dst.at(0, y_idx, 3);
    local_tmp[4] = dev_speeds_dst.at(0, y_idx, 4);
    local_tmp[6] = dev_speeds_dst.at(0, y_idx, 6);
    local_tmp[7] = dev_speeds_dst.at(0, y_idx, 7);
    const float inlets = dev_inlets[y_idx];
    const float local_denisty = ((local_tmp[0] + local_tmp[2] + local_tmp[4]) + 2.f * (local_tmp[3] + local_tmp[6] + local_tmp[7])) / (1.f - inlets);
    dev_speeds_dst.at(0, y_idx, 1) = local_tmp[3] + cst1 * local_denisty * inlets;
    dev_speeds_dst.at(0, y_idx, 5) = local_tmp[7] - cst3 * (local_tmp[2] - local_tmp[4]) + cst2 * local_denisty * inlets;
    dev_speeds_dst.at(0, y_idx, 8) = local_tmp[6] + cst3 * (local_tmp[2] - local_tmp[4]) + cst2 * local_denisty * inlets;
}

__device__ void stream(float* tmp, const u32 x_idx, const u32 y_idx, const u32 nx, const u32 ny, array3d<float, Device> dev_speeds_dst) {
    
    dev_speeds_dst.at(x_idx, y_idx, 0) = tmp[0];

    if(x_idx + 1 < nx) {
        dev_speeds_dst.at(x_idx + 1, y_idx, 1) = tmp[1];
        if(y_idx + 1 < ny) {
            dev_speeds_dst.at(x_idx + 1, y_idx + 1, 5) = tmp[5];
        }
    }

    if(y_idx + 1 < ny) {
        dev_speeds_dst.at(x_idx, y_idx + 1, 2) = tmp[2];
        if(x_idx > 0) {
            dev_speeds_dst.at(x_idx - 1, y_idx + 1, 6) = tmp[6];
        }
    } else {
        //y_idx + 1 == ny -> top wall
        dev_speeds_dst.at(x_idx, y_idx, 4) = tmp[2];
        dev_speeds_dst.at(x_idx, y_idx, 7) = tmp[5];
        dev_speeds_dst.at(x_idx, y_idx, 8) = tmp[6];
    }

    if(x_idx > 0) {
        dev_speeds_dst.at(x_idx - 1, y_idx, 3) = tmp[3];
        if(y_idx > 0) {
            dev_speeds_dst.at(x_idx - 1, y_idx - 1, 7) = tmp[7];
        }
    }

    if(y_idx > 0) {
        dev_speeds_dst.at(x_idx, y_idx - 1, 4) = tmp[4];
        if(x_idx + 1 < nx) {
            dev_speeds_dst.at(x_idx + 1, y_idx - 1, 8) = tmp[8];
        }
    } else {
        //y_idx == 0 -> bottom wall
        dev_speeds_dst.at(x_idx, y_idx, 2) = tmp[4];
        dev_speeds_dst.at(x_idx, y_idx, 5) = tmp[7];
        dev_speeds_dst.at(x_idx, y_idx, 6) = tmp[8];
    }

    
}

__device__ void boundary(float* tmp, const u32 x_idx, const u32 y_idx, const u32 nx, const u32 ny, array3d<float, Device> dev_speeds_dst, const float* dev_inlets) {

    //top wall
    if(y_idx == ny - 1) {
        dev_speeds_dst.at(x_idx, y_idx, 4) = tmp[2];
        dev_speeds_dst.at(x_idx, y_idx, 7) = tmp[5];
        dev_speeds_dst.at(x_idx, y_idx, 8) = tmp[6];
    }
    //bottom wall
    if(y_idx == 0) {
        dev_speeds_dst.at(x_idx, y_idx, 2) = tmp[4];
        dev_speeds_dst.at(x_idx, y_idx, 5) = tmp[7];
        dev_speeds_dst.at(x_idx, y_idx, 6) = tmp[8];
    }

    constexpr float cst1 = 2.f / 3.f;
    constexpr float cst2 = 1.f / 6.f;
    constexpr float cst3 = 1.f / 2.f;

    //left wall
    if(x_idx == 0) {
        tmp[0] = dev_speeds_dst.at(0, y_idx, 0);
        tmp[2] = dev_speeds_dst.at(0, y_idx, 2);
        tmp[3] = dev_speeds_dst.at(0, y_idx, 3);
        tmp[4] = dev_speeds_dst.at(0, y_idx, 4);
        tmp[6] = dev_speeds_dst.at(0, y_idx, 6);
        tmp[7] = dev_speeds_dst.at(0, y_idx, 7);
        const float inlets = dev_inlets[y_idx];
        const float local_density = ((tmp[0] + tmp[2] + tmp[4]) + 2.f * (tmp[3] + tmp[6] + tmp[7])) / (1.f - inlets);
        dev_speeds_dst.at(0, y_idx, 1) = tmp[3] + cst1 * local_density * inlets;
        dev_speeds_dst.at(0, y_idx, 5) = tmp[7] - cst3 * (tmp[2] - tmp[4]) + cst2 * local_density * inlets;
        dev_speeds_dst.at(0, y_idx, 8) = tmp[6] + cst3 * (tmp[2] - tmp[4]) + cst2 * local_density * inlets;
    }

    //right wall
    if(x_idx == nx - 1) {
        dev_speeds_dst.at(x_idx, y_idx, 0) = dev_speeds_dst.at(x_idx - 1, y_idx, 0);
        dev_speeds_dst.at(x_idx, y_idx, 1) = dev_speeds_dst.at(x_idx - 1, y_idx, 1);
        dev_speeds_dst.at(x_idx, y_idx, 2) = dev_speeds_dst.at(x_idx - 1, y_idx, 2);
        dev_speeds_dst.at(x_idx, y_idx, 3) = dev_speeds_dst.at(x_idx - 1, y_idx, 3);
        dev_speeds_dst.at(x_idx, y_idx, 4) = dev_speeds_dst.at(x_idx - 1, y_idx, 4);
        dev_speeds_dst.at(x_idx, y_idx, 5) = dev_speeds_dst.at(x_idx - 1, y_idx, 5);
        dev_speeds_dst.at(x_idx, y_idx, 6) = dev_speeds_dst.at(x_idx - 1, y_idx, 6);
        dev_speeds_dst.at(x_idx, y_idx, 7) = dev_speeds_dst.at(x_idx - 1, y_idx, 7);
        dev_speeds_dst.at(x_idx, y_idx, 8) = dev_speeds_dst.at(x_idx - 1, y_idx, 8);
    }
}

__global__ void _d2q9_bgk(t_param params, gpu_data_t gpu_data) {
    float tmp[9]; //locate in on-chip memory

    for(u32 i = 0 ; i < params.maxIters ; ++i) {
        for(u32 y_idx = blockIdx.x ; y_idx < params.ny ; y_idx += gridDim.x) { 
            for(u32 x_idx = threadIdx.x ; x_idx < params.nx ; x_idx += blockDim.x) {
                load_speeds(x_idx, y_idx, tmp, gpu_data.dev_speed_src);
                if(check_obstacle(gpu_data.obstacles_pitch, x_idx, y_idx)) {
                    obstacle(tmp);
                } else {
                    collision(tmp, params.omega);
                }
                save_speeds(x_idx, y_idx, params.nx, params.ny, tmp, gpu_data.dev_speed_dst);
            }
        }
        this_grid().sync();
        if(threadIdx.x == 0) {
            for(u32 y_idx = blockIdx.x ; y_idx < params.ny ; y_idx += gridDim.x) {
                left_boundary(y_idx, gpu_data.dev_speed_dst, tmp, gpu_data.dev_inlets);
            }
        }
        

        auto tmp = gpu_data.dev_speed_src._ptr.ptr;
        gpu_data.dev_speed_src._ptr.ptr = gpu_data.dev_speed_dst._ptr.ptr;
        gpu_data.dev_speed_dst._ptr.ptr =  tmp;

    }
}

__global__ void _init_device_speeds(array3d<float, Device> dev_speeds, float density, const u32 nx, const u32 ny) {
    for(u32 y_idx = blockIdx.x ; y_idx < ny ; y_idx += gridDim.x) {
        for(u32 x_idx = threadIdx.x ; x_idx < nx ; x_idx += blockDim.x) {
            dev_speeds.at(x_idx, y_idx, 0) = density * 4.f / 9.f;
            dev_speeds.at(x_idx, y_idx, 1) = density / 9.f;
            dev_speeds.at(x_idx, y_idx, 2) = density / 9.f;
            dev_speeds.at(x_idx, y_idx, 3) = density / 9.f;
            dev_speeds.at(x_idx, y_idx, 4) = density / 9.f;
            dev_speeds.at(x_idx, y_idx, 5) = density / 36.f;
            dev_speeds.at(x_idx, y_idx, 6) = density / 36.f;
            dev_speeds.at(x_idx, y_idx, 7) = density / 36.f;
            dev_speeds.at(x_idx, y_idx, 8) = density / 36.f;
        }
    }
}

__global__ void _init_device_inlets(u32 ny, float velocity, bool type, float* dev_inlets) {
    for(u32 y_idx = blockDim.x * blockIdx.x + threadIdx.x ; y_idx < ny ; y_idx += gridDim.x * blockDim.x) {
        dev_inlets[y_idx] = type ? velocity * 4.f * (1.f - (float)y_idx / (float)ny) * ((float)y_idx + 1.f) / (float)ny : velocity;
    }
}

__host__ void init_device_data(const t_param& params, gpu_data_t& gpu_data,  const char* obstaclesfile, hipStream_t stream = 0) {
    gpu_data.obstacles_pitch = (params.nx - 1) / 32 + 1;
    if(gpu_data.obstacles_pitch * params.ny > 0x4000) {
        die("don't have enough constant memory!", __LINE__, __FILE__);
    }
    auto_launch_kernel_1D(_init_device_speeds, 0, stream, gpu_data.dev_speed_src, params.density, params.nx, params.ny);
    auto_launch_kernel_1D(_init_device_inlets, 0, stream, params.ny, params.velocity, params.type, gpu_data.dev_inlets);
    std::ifstream obstacle_file{obstaclesfile};
    std::string line;
    int nx, ny, block;

    if(!obstacle_file) {
        die(std::string("Could not open the obstacle file:") + obstaclesfile, __LINE__, __FILE__);
    }

    while(std::getline(obstacle_file, line)) {
        if(sscanf(line.data(), "%d %d %d", &nx, &ny, &block) != 3) {
            die("expected 3 values per line in obstacle file", __LINE__, __FILE__);
        }

        ny = ny + params.ny / 2;

        if(nx < 0 || nx >= params.nx) die("obstacle x-coord out of range", __LINE__, __FILE__);
        if(ny < 0 || ny >= params.ny) die("obstacle y-coord out of range", __LINE__, __FILE__);
        if(block != 1) die("obstacle blocked value should be 1", __LINE__, __FILE__);
        set_obstacle(gpu_data.obstacles_pitch, nx, ny);
    }

    CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_obstacles), host_obstacles, sizeof(dev_obstacles), 0, hipMemcpyDefault, stream));
}

__host__ void copy_device_speeds_to_host(const t_param& params, array3d<float, Device> dev_speeds, float* host_speeds, hipStream_t stream = 0) {
    hipMemcpy3DParms cpy_param {0};
    cpy_param.srcPtr = dev_speeds.get();
    cpy_param.dstPtr = make_hipPitchedPtr(host_speeds, sizeof(float) * params.nx, params.nx, params.ny);
    cpy_param.extent = hipExtent{sizeof(float) * params.nx, params.ny, 9};
    cpy_param.kind = hipMemcpyDeviceToHost; 
    CUDA_CALL(hipMemcpy3DAsync(&cpy_param, stream));
}

__host__ void print_host_speeds(const t_param& params, float* host_speeds) {
    for(int i = 0 ; i < 9 ; ++i) {
        printf("speeds: %d\n", i);
        for(int yy = params.ny - 1 ; yy >= 0 ; --yy) {
            for(int xx = 0 ; xx < params.nx ; ++xx) {
                printf("%.5f ", host_speeds[xx + (yy + params.ny * i) * params.nx]);
            }
            printf("\n");
        }
    }
}

__host__ void write_state(const char* output_path, const t_param& params, float* host_speeds, u32 obstacles_pitch) {
    std::string out_file_name {output_path};
    out_file_name += "/gpu_final_state.dat";
    std::ofstream out {out_file_name};
    if(!out) {
        die("Could not open the output file\n", __LINE__, __FILE__);
    }
    float u_x, u_y, u, local_density;
    for(int jj = 0 ; jj < params.ny ; ++jj) {
        for(int ii = 0 ; ii < params.nx ; ++ii) {
            if(check_obstacle_host(obstacles_pitch, ii, jj)) {
                u = -0.05f;
            } else {
                local_density = 0.f;
                for(int kk = 0 ; kk < 9 ; ++kk) {
                    local_density += host_speeds[ii + (jj + params.ny * kk) * params.nx];
                }
                u_x =  ((
                        host_speeds[ii + (jj + params.ny * 1) * params.nx] + 
                        host_speeds[ii + (jj + params.ny * 5) * params.nx] + 
                        host_speeds[ii + (jj + params.ny * 8) * params.nx]
                       ) - (
                        host_speeds[ii + (jj + params.ny * 3) * params.nx] + 
                        host_speeds[ii + (jj + params.ny * 6) * params.nx] + 
                        host_speeds[ii + (jj + params.ny * 7) * params.nx]
                       )) / local_density;
                u_y = ((
                       host_speeds[ii + (jj + params.ny * 2) * params.nx] + 
                       host_speeds[ii + (jj + params.ny * 5) * params.nx] + 
                       host_speeds[ii + (jj + params.ny * 6) * params.nx]
                      ) - (
                        host_speeds[ii + (jj + params.ny * 4) * params.nx] + 
                        host_speeds[ii + (jj + params.ny * 7) * params.nx] + 
                        host_speeds[ii + (jj + params.ny * 8) * params.nx]
                      )) / local_density;
                u = sqrtf(u_x * u_x + u_y * u_y);
                out << ii << " " << jj << " " << std::setprecision(12) << std::fixed << u << std::endl;
            }
        }
    }
}

inline void usage(const char* exe) {
  fprintf(stderr, "Usage: %s <paramfile> <obstaclefile> <output_directory>\n", exe);
  exit(1);
}

int main(int argc, char *argv[]) {
    char*    paramfile = nullptr;    /* name of the input parameter file */
    char*    obstaclefile = nullptr; /* name of a the input obstacle file */
    char*    out_dir = nullptr;      /* name of output directory */
    t_param  params;                 /* struct to hold parameter values */
    gpu_data_t gpu_data;
    hipStream_t hip_stream;
    CUDA_CALL(hipStreamCreate(&hip_stream));

    //handle input
    if(argc != 4) {
        usage(argv[0]);
    }

    paramfile = argv[1];
    obstaclefile = argv[2];
    out_dir = argv[3];
    //handle input end
    

    load_params(params, paramfile);
    array3d<float, Host> p_host_1 {params.nx, params.ny, 9};
    array3d<float, Host> p_host_2 {params.nx, params.ny, 9};
    float *host_speeds;
    CUDA_CALL(hipHostMalloc(&host_speeds, sizeof(float) * 9 * params.nx * params.ny));

    gpu_data.dev_speed_src = p_host_1;
    gpu_data.dev_speed_dst = p_host_2;
    hipEvent_t init_start, init_end;
    hipEvent_t cal_start, cal_end;
    float init_elapsedTime;
    float cal_elapsedTime;
    CUDA_CALL(hipEventCreate(&init_start));
    CUDA_CALL(hipEventCreate(&init_end));
    CUDA_CALL(hipEventCreate(&cal_start));
    CUDA_CALL(hipEventCreate(&cal_end));
    CUDA_CALL(hipEventRecord(init_start, hip_stream));
    CUDA_CALL(hipMallocAsync(&gpu_data.dev_inlets, params.ny * sizeof(float), hip_stream));
    init_device_data(params, gpu_data, obstaclefile, hip_stream);
    CUDA_CALL(hipEventRecord(init_end, hip_stream));
    CUDA_CALL(hipEventRecord(cal_start, hip_stream));
    auto_launch_kernel_1D(_d2q9_bgk, 0, hip_stream, params, gpu_data);
    cuda_error_check("_d2q9_bgk", __LINE__, __FILE__);
    if(params.maxIters % 2 == 0) {
        copy_device_speeds_to_host(params, p_host_1, host_speeds, hip_stream);
    } else {
        copy_device_speeds_to_host(params, p_host_2, host_speeds, hip_stream);
    }
    CUDA_CALL(hipEventRecord(cal_end, hip_stream));


    CUDA_CALL(hipStreamSynchronize(hip_stream));
    write_state(out_dir, params, host_speeds, gpu_data.obstacles_pitch);
    printf("==done==\n");
    CUDA_CALL(hipEventElapsedTime(&init_elapsedTime, init_start, init_end));
    CUDA_CALL(hipEventElapsedTime(&cal_elapsedTime, cal_start, cal_end));
    printf("Elapsed Init time:\t\t\t%.5f (ms)\n",    init_elapsedTime);
    printf("Elapsed Compute time:\t\t\t%.5f (ms)\n", cal_elapsedTime);
    CUDA_CALL(hipStreamDestroy(hip_stream));
    //print_host_speeds(params, host_speeds);
    CUDA_CALL(hipHostFree(host_speeds));
    CUDA_CALL(hipFree(gpu_data.dev_inlets));
    return 0;
}